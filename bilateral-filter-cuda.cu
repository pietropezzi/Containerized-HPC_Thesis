#include "hip/hip_runtime.h"
/* Pietro Pezzi - pietro.pezzi3@studio.unibo.it - 0000925022 */
#include "hpc.h"
#include "ppm.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define PI 3.142857

#define BLKDIM 16
#define RADIUS 10

typedef struct {
    double red, green, blue;
} RGB_Double;

// Sets all parameters of given RGB_Double with the given value.
__device__ void set_dpixel(RGB_Double *pixel, double value)
{
    pixel->blue = value;
    pixel->green = value;
    pixel->red = value;
}

/* Simplifies indexing on a M*N grid. */
__device__ int IDX(int i, int j, int M, int N)
{
    /* wrap-around */
    i = (i + M) % M;
    j = (j + N) % N;
    return i * N + j;
}
__device__ double gauss_kernel(int x, double sigma)
{
    return (1 / (sqrt((double)(2.0 * PI)) * sigma)) *
           exp((double)(-(x * x) / (2.0 * (sigma * sigma))));
}

/* Returns the eucliedean distance between two points (x1,y1) and (x2,y2). */
__device__ double eucl_dist(int x1, int y1, int x2, int y2)
{
    return sqrt((double)((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1)));
}

/* Kernel function to apply bilateral filtering algorithm on NxN image.
 *
 * d_imgin is the RGB_Pixel pointer to the original image in the GPU global
 * memory. 
 * d_imgout is the RGB_Pixel pointer to the RGB_Pixel array where the
 * result of the filter will be stored, in the GPU Global memory. 
 * N in image's width/height. 
 * The parameter sigma_color determines the range of values that
 * will have a considerable weight when computing the new pixel value, given
 * that the value of a pixel is \in [0, 255] sigma_color should be set
 * accordingly. 
 * The parameter sigma_spatial determines the range of distance
 * that will have a considerable weight when computing the new pixel value. The
 * parameter filtered is the RGB_Pixel pointer where the new pixel's values will
 * be stored.
 */
__global__ void bilateral_filter(RGB_Pixel *d_imgin, RGB_Pixel *d_imgout, int N,
                                 int sigma_color, int sigma_spatial)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int local_x = threadIdx.x + RADIUS;
    const int local_y = threadIdx.y + RADIUS;
    int i, j;

    const int dim = BLKDIM + (RADIUS * 2);
    __shared__ RGB_Pixel local_array[dim * dim];

    // Load local_array.
    // load "personal" pixel.
    local_array[IDX(local_x, local_y, dim, dim)] = d_imgin[IDX(x, y, N, N)];

    // Load "sides" ghost pixels.
    if (local_x == RADIUS) {
        for (i = 1; i <= RADIUS; i++) {
            local_array[IDX(local_x - i, local_y, dim, dim)] =
                d_imgin[IDX(x - i, y, N, N)];
        }
    }
    if (local_x == RADIUS + blockDim.x - 1) {
        for (i = 1; i <= RADIUS; i++) {
            local_array[IDX(local_x + i, local_y, dim, dim)] =
                d_imgin[IDX(x + i, y, N, N)];
        }
    }
    if (local_y == RADIUS) {
        for (i = 1; i <= RADIUS; i++) {
            local_array[IDX(local_x, local_y - i, dim, dim)] =
                d_imgin[IDX(x, y - i, N, N)];
        }
    }
    if (local_y == RADIUS + blockDim.y - 1) {
        for (i = 1; i <= RADIUS; i++) {
            local_array[IDX(local_x, local_y + i, dim, dim)] =
                d_imgin[IDX(x, y + i, N, N)];
        }
    }

    // Load "corner" ghost pixels.
    if (local_x == RADIUS && local_y == RADIUS) {
        for (i = 1; i <= RADIUS; i++) {
            for (j = 1; j <= RADIUS; j++) {
                local_array[IDX(local_x - i, local_y - j, dim, dim)] =
                    d_imgin[IDX(x - i, y - j, N, N)];
            }
        }
    }
    if (local_x == RADIUS && local_y == RADIUS + BLKDIM - 1) {
        for (i = 1; i <= RADIUS; i++) {
            for (j = 1; j <= RADIUS; j++) {
                local_array[IDX(local_x - i, local_y + j, dim, dim)] =
                    d_imgin[IDX(x - i, y + j, N, N)];
            }
        }
    }
    if (local_x == RADIUS + BLKDIM - 1 && local_y == RADIUS) {
        for (i = 1; i <= RADIUS; i++) {
            for (j = 1; j <= RADIUS; j++) {
                local_array[IDX(local_x + i, local_y - j, dim, dim)] =
                    d_imgin[IDX(x + i, y - j, N, N)];
            }
        }
    }
    if (local_x == RADIUS + BLKDIM - 1 && local_y == RADIUS + BLKDIM - 1) {
        for (i = 1; i <= RADIUS; i++) {
            for (j = 1; j <= RADIUS; j++) {
                local_array[IDX(local_x + i, local_y + j, dim, dim)] =
                    d_imgin[IDX(x + i, y + j, N, N)];
            }
        }
    }
    __syncthreads();

    // Apply Bilateral filter on pixel.
    RGB_Double total_weight, partial_weight, neighborhood, gauss_color;
    double gauss_spatial;

    set_dpixel(&total_weight, 0.0);
    set_dpixel(&partial_weight, 0.0);
    set_dpixel(&neighborhood, 0.0);
    for (i = -RADIUS; i <= RADIUS; i++) {
        for (j = -RADIUS; j <= RADIUS; j++) {
            gauss_color.red = gauss_kernel(
                local_array[IDX(local_x + i, local_y + j, dim, dim)].red -
                    local_array[IDX(local_x, local_y, dim, dim)].red,
                sigma_color);
            gauss_color.green = gauss_kernel(
                local_array[IDX(local_x + i, local_y + j, dim, dim)].green -
                    local_array[IDX(local_x, local_y, dim, dim)].green,
                sigma_color);
            gauss_color.blue = gauss_kernel(
                local_array[IDX(local_x + i, local_y + j, dim, dim)].blue -
                    local_array[IDX(local_x, local_y, dim, dim)].blue,
                sigma_color);
            gauss_spatial = gauss_kernel(
                eucl_dist(local_x + i, local_y + j, local_x, local_y),
                sigma_spatial);
            partial_weight.red = gauss_color.red * gauss_spatial;
            partial_weight.green = gauss_color.green * gauss_spatial;
            partial_weight.blue = gauss_color.blue * gauss_spatial;
            neighborhood.red +=
                (local_array[IDX(local_x + i, local_y + j, dim, dim)].red *
                 partial_weight.red);
            neighborhood.green +=
                (local_array[IDX(local_x + i, local_y + j, dim, dim)].green *
                 partial_weight.green);
            neighborhood.blue +=
                (local_array[IDX(local_x + i, local_y + j, dim, dim)].blue *
                 partial_weight.blue);
            total_weight.red += partial_weight.red;
            total_weight.green += partial_weight.green;
            total_weight.blue += partial_weight.blue;
        }
    }

    // Update pixel value in global result array.
    d_imgout[IDX(x, y, N, N)].red = (int)(neighborhood.red / total_weight.red);
    d_imgout[IDX(x, y, N, N)].green =
        (int)(neighborhood.green / total_weight.green);
    d_imgout[IDX(x, y, N, N)].blue =
        (int)(neighborhood.blue / total_weight.blue);
}

int main(int argc, char *argv[])
{
    FILE *in = NULL;
    PPM_Image *img = NULL;
    double sigma_color, sigma_spatial;
    RGB_Pixel *d_imgin, *d_imgout;
    double t1, t2;

    if (argc != 4) {
        fprintf(stderr, "USAGE: <imagepath> <sigma_color> <sigma_spatial>.\n");
        return 1;
    }
    if ((in = fopen(argv[1], "rb")) == NULL) {
        fprintf(stderr, "FATAL: Image file '%s' not found.\n", argv[1]);
        return 1;
    }
    if (sscanf(argv[2], "%lf", &sigma_color) != 1) {
        fprintf(stderr,
                "FATAL: Error while reading the sigma-color's value.\n");
        fclose(in);
        return 1;
    }
    if (sscanf(argv[3], "%lf", &sigma_spatial) != 1) {
        fprintf(stderr,
                "FATAL: Error while reading the sigma-spatial's value.\n");
        fclose(in);
        return 1;
    }

    img = (PPM_Image *)malloc(sizeof(PPM_Image));
    assert(img != NULL);
    if (load_PPM(in, img) == 1) {
        fprintf(stderr, "FATAL: Error while loading image.\n");
        fclose(in);
        return 1;
    }
    fclose(in);

    if (img->header.height != img->header.width) {
        fprintf(stderr, "FATAL: Image's width and height must be equal.\n");
        free(img);
        return 1;
    }
    if (img->header.height % BLKDIM != 0) {
        fprintf(stderr,
                "FATAL: Image's height and width must be a multiple of %d.\n",
                BLKDIM);
        free(img);
        return 1;
    }

    const dim3 block(BLKDIM, BLKDIM);
    const dim3 grid((img->header.height + BLKDIM - 1) / BLKDIM,
                    (img->header.height + BLKDIM - 1) / BLKDIM);

    const size_t img_size =
        (img->header.height * img->header.width) * sizeof(RGB_Pixel);
    cudaSafeCall(hipMalloc((void **)&d_imgin, img_size));
    cudaSafeCall(hipMalloc((void **)&d_imgout, img_size));
    cudaSafeCall(
        hipMemcpy(d_imgin, img->pixelArray, img_size, hipMemcpyHostToDevice));

    t1 = hpc_gettime();
    bilateral_filter<<<grid, block>>>(d_imgin, d_imgout, img->header.width,
                                      sigma_color, sigma_spatial);
    cudaCheckError();
    hipDeviceSynchronize();
    t2 = hpc_gettime();

    cudaSafeCall(hipMemcpy(img->pixelArray, d_imgout, img_size,
                            hipMemcpyDeviceToHost));

    if (write_PPM("filtered_image_cuda.ppm", img) == 1) {
        hipFree(d_imgin);
        hipFree(d_imgout);
        free(img);
        return 1;
    }

    printf("Cuda imp execution time: %f\n", t2-t1);
    hipFree(d_imgin);
    hipFree(d_imgout);
    free(img);
    return 0;
}
